#include <hip/hip_runtime.h>


#include "sm_30_intrinsics.h"

#include <stdio.h>
#include <memory.h>
#include <stdint.h>

// aus cpu-miner.c
extern int device_map[8];

// diese Struktur wird in der Init Funktion angefordert
static hipDeviceProp_t props[8];

static uint32_t *d_tempBranch1Nonces[8];
static uint32_t *d_numValid[8];
static uint32_t *h_numValid[8];

static uint32_t *d_partSum[2][8]; // f�r bis zu vier partielle Summen

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

// True/False tester
typedef uint32_t(*cuda_compactTestFunction_t)(uint32_t *inpHash);

__device__ uint32_t QuarkTrueTest(uint32_t *inpHash)
{
	return ((inpHash[0] & 0x08) == 0x08);
}

__device__ uint32_t QuarkFalseTest(uint32_t *inpHash)
{
	return ((inpHash[0] & 0x08) == 0);
}

__device__ cuda_compactTestFunction_t d_QuarkTrueFunction = QuarkTrueTest, d_QuarkFalseFunction = QuarkFalseTest;
cuda_compactTestFunction_t h_QuarkTrueFunction[8], h_QuarkFalseFunction[8];

// Setup-Funktionen
__host__ void quark_compactTest_cpu_init(int thr_id, int threads)
{
	hipGetDeviceProperties(&props[thr_id], device_map[thr_id]);

	hipMemcpyFromSymbol(&h_QuarkTrueFunction[thr_id], HIP_SYMBOL(d_QuarkTrueFunction), sizeof(cuda_compactTestFunction_t));
	hipMemcpyFromSymbol(&h_QuarkFalseFunction[thr_id], HIP_SYMBOL(d_QuarkFalseFunction), sizeof(cuda_compactTestFunction_t));

	// wir brauchen auch Speicherplatz auf dem Device
	hipMalloc(&d_tempBranch1Nonces[thr_id], sizeof(uint32_t) * threads * 2);	
	hipMalloc(&d_numValid[thr_id], 2*sizeof(uint32_t));
	hipHostMalloc(&h_numValid[thr_id], 2*sizeof(uint32_t), hipHostMallocDefault);

	uint32_t s1;
	s1 = (threads / 256) * 2;

	hipMalloc(&d_partSum[0][thr_id], sizeof(uint32_t) * s1); // BLOCKSIZE (Threads/Block)
	hipMalloc(&d_partSum[1][thr_id], sizeof(uint32_t) * s1); // BLOCKSIZE (Threads/Block)
}

#if __CUDA_ARCH__ < 300
/**
 * __shfl_up() calculates a source lane ID by subtracting delta from the caller's lane ID, and clamping to the range 0..width-1
 */
#undef __shfl_up
#define __shfl_up(var, delta, width) (0)
#endif

// Die Summenfunktion (vom NVIDIA SDK)
__global__ void quark_compactTest_gpu_SCAN(uint32_t *data, int width, uint32_t *partial_sums=NULL, cuda_compactTestFunction_t testFunc=NULL, int threads=0, uint32_t startNounce=0, uint32_t *inpHashes=NULL, uint32_t *d_validNonceTable=NULL)
{
	extern __shared__ uint32_t sums[];
	int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
	//int lane_id = id % warpSize;
	int lane_id = id % width;
	// determine a warp_id within a block
	 //int warp_id = threadIdx.x / warpSize;
	int warp_id = threadIdx.x / width;

	sums[lane_id] = 0;

	// Below is the basic structure of using a shfl instruction
	// for a scan.
	// Record "value" as a variable - we accumulate it along the way
	uint32_t value;
	if(testFunc != NULL)
	{
		if (id < threads)
		{
			uint32_t *inpHash;
			if(d_validNonceTable == NULL)
			{
				// keine Nonce-Liste
				inpHash = &inpHashes[id<<4];
			}else
			{
				// Nonce-Liste verf�gbar
				int nonce = d_validNonceTable[id] - startNounce;
				inpHash = &inpHashes[nonce<<4];
			}			
			value = (*testFunc)(inpHash);
		}else
		{
			value = 0;
		}
	}else
	{
		value = data[id];
	}

	__syncthreads();

	// Now accumulate in log steps up the chain
	// compute sums, with another thread's value who is
	// distance delta away (i).  Note
	// those threads where the thread 'i' away would have
	// been out of bounds of the warp are unaffected.  This
	// creates the scan sum.
#pragma unroll

	for (int i=1; i<=width; i*=2)
	{
		uint32_t n = __shfl_up((int)value, i, width);

		if (lane_id >= i) value += n;
	}

	// value now holds the scan value for the individual thread
	// next sum the largest values for each warp

	// write the sum of the warp to smem
	//if (threadIdx.x % warpSize == warpSize-1)
	if (threadIdx.x % width == width-1)
	{
		sums[warp_id] = value;
	}

	__syncthreads();

	//
	// scan sum the warp sums
	// the same shfl scan operation, but performed on warp sums
	//
	if (warp_id == 0)
	{
		uint32_t warp_sum = sums[lane_id];

		for (int i=1; i<=width; i*=2)
		{
			uint32_t n = __shfl_up((int)warp_sum, i, width);

		if (lane_id >= i) warp_sum += n;
		}

		sums[lane_id] = warp_sum;
	}

	__syncthreads();

	// perform a uniform add across warps in the block
	// read neighbouring warp's sum and add it to threads value
	uint32_t blockSum = 0;

	if (warp_id > 0)
	{
		blockSum = sums[warp_id-1];
	}

	value += blockSum;

	// Now write out our result
	data[id] = value;

	// last thread has sum, write write out the block's sum
	if (partial_sums != NULL && threadIdx.x == blockDim.x-1)
	{
		partial_sums[blockIdx.x] = value;
	}
}

// Uniform add: add partial sums array
__global__ void quark_compactTest_gpu_ADD(uint32_t *data, uint32_t *partial_sums, int len)
{
	__shared__ uint32_t buf;
	int id = ((blockIdx.x * blockDim.x) + threadIdx.x);

	if (id > len) return;

	if (threadIdx.x == 0)
	{
		buf = partial_sums[blockIdx.x];
	}

	__syncthreads();
	data[id] += buf;
}

// Der Scatter
__global__ void quark_compactTest_gpu_SCATTER(uint32_t *sum, uint32_t *outp, cuda_compactTestFunction_t testFunc, int threads=0, uint32_t startNounce=0, uint32_t *inpHashes=NULL, uint32_t *d_validNonceTable=NULL)
{
	int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
	uint32_t actNounce = id;
	uint32_t value;
	if (id < threads)
	{
//		uint32_t nounce = startNounce + id;
		uint32_t *inpHash;
		if(d_validNonceTable == NULL)
		{
			// keine Nonce-Liste
			inpHash = &inpHashes[id<<4];
		}else
		{
			// Nonce-Liste verf�gbar
			int nonce = d_validNonceTable[id] - startNounce;
			actNounce = nonce;
			inpHash = &inpHashes[nonce<<4];
		}

		value = (*testFunc)(inpHash);
	}else
	{
		value = 0;
	}

	if( value )
	{
		int idx = sum[id];
		if(idx > 0)
			outp[idx-1] = startNounce + actNounce;
	}
}

__host__ static uint32_t quark_compactTest_roundUpExp(uint32_t val)
{
	if(val == 0)
		return 0;

	uint32_t mask = 0x80000000;
	while( (val & mask) == 0 ) mask = mask >> 1;

	if( (val & (~mask)) != 0 )
		return mask << 1;

	return mask;
}

__host__ void quark_compactTest_cpu_singleCompaction(int thr_id, int threads, uint32_t *nrm,
														uint32_t *d_nonces1, cuda_compactTestFunction_t function,
														uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable)
{
	int orgThreads = threads;
	threads = (int)quark_compactTest_roundUpExp((uint32_t)threads);
	// threadsPerBlock ausrechnen
	int blockSize = 256;
	int nSummen = threads / blockSize;

	int thr1 = (threads+blockSize-1) / blockSize;
	int thr2 = threads / (blockSize*blockSize);
	int blockSize2 = (nSummen < blockSize) ? nSummen : blockSize;
	int thr3 = (nSummen + blockSize2-1) / blockSize2;

	bool callThrid = (thr2 > 0) ? true : false;

	// Erster Initialscan
	quark_compactTest_gpu_SCAN<<<thr1,blockSize, 32*sizeof(uint32_t)>>>(
		d_tempBranch1Nonces[thr_id], 32, d_partSum[0][thr_id], function, orgThreads, startNounce, inpHashes, d_validNonceTable);	

	// weitere Scans
	if(callThrid)
	{		
		quark_compactTest_gpu_SCAN<<<thr2,blockSize, 32*sizeof(uint32_t)>>>(d_partSum[0][thr_id], 32, d_partSum[1][thr_id]);
		quark_compactTest_gpu_SCAN<<<1, thr2, 32*sizeof(uint32_t)>>>(d_partSum[1][thr_id], (thr2>32) ? 32 : thr2);
	}else
	{
		quark_compactTest_gpu_SCAN<<<thr3,blockSize2, 32*sizeof(uint32_t)>>>(d_partSum[0][thr_id], (blockSize2>32) ? 32 : blockSize2);
	}

	// Sync + Anzahl merken
	hipStreamSynchronize(NULL);

	if(callThrid)
		hipMemcpy(nrm, &(d_partSum[1][thr_id])[thr2-1], sizeof(uint32_t), hipMemcpyDeviceToHost);
	else
		hipMemcpy(nrm, &(d_partSum[0][thr_id])[nSummen-1], sizeof(uint32_t), hipMemcpyDeviceToHost);

	
	// Addieren
	if(callThrid)
	{
		quark_compactTest_gpu_ADD<<<thr2-1, blockSize>>>(d_partSum[0][thr_id]+blockSize, d_partSum[1][thr_id], blockSize*thr2);
	}
	quark_compactTest_gpu_ADD<<<thr1-1, blockSize>>>(d_tempBranch1Nonces[thr_id]+blockSize, d_partSum[0][thr_id], threads);
	
	// Scatter
	quark_compactTest_gpu_SCATTER<<<thr1,blockSize,0>>>(d_tempBranch1Nonces[thr_id], d_nonces1, 
		function, orgThreads, startNounce, inpHashes, d_validNonceTable);

	// Sync
	hipStreamSynchronize(NULL);
}

////// ACHTUNG: Diese funktion geht aktuell nur mit threads > 65536 (Am besten 256 * 1024 oder 256*2048)
__host__ void quark_compactTest_cpu_dualCompaction(int thr_id, int threads, uint32_t *nrm,
													 uint32_t *d_nonces1, uint32_t *d_nonces2,
													 uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable)
{
	quark_compactTest_cpu_singleCompaction(thr_id, threads, &nrm[0], d_nonces1, h_QuarkTrueFunction[thr_id], startNounce, inpHashes, d_validNonceTable);
	quark_compactTest_cpu_singleCompaction(thr_id, threads, &nrm[1], d_nonces2, h_QuarkFalseFunction[thr_id], startNounce, inpHashes, d_validNonceTable);

	/*
	// threadsPerBlock ausrechnen
	int blockSize = 256;
	int thr1 = threads / blockSize;
	int thr2 = threads / (blockSize*blockSize);

	// 1
	quark_compactTest_gpu_SCAN<<<thr1,blockSize, 32*sizeof(uint32_t)>>>(d_tempBranch1Nonces[thr_id], 32, d_partSum1[thr_id], h_QuarkTrueFunction[thr_id], threads, startNounce, inpHashes);
	quark_compactTest_gpu_SCAN<<<thr2,blockSize, 32*sizeof(uint32_t)>>>(d_partSum1[thr_id], 32, d_partSum2[thr_id]);
	quark_compactTest_gpu_SCAN<<<1, thr2, 32*sizeof(uint32_t)>>>(d_partSum2[thr_id], (thr2>32) ? 32 : thr2);
	cudaStreamSynchronize(NULL);
	cudaMemcpy(&nrm[0], &(d_partSum2[thr_id])[thr2-1], sizeof(uint32_t), cudaMemcpyDeviceToHost);
	quark_compactTest_gpu_ADD<<<thr2-1, blockSize>>>(d_partSum1[thr_id]+blockSize, d_partSum2[thr_id], blockSize*thr2);
	quark_compactTest_gpu_ADD<<<thr1-1, blockSize>>>(d_tempBranch1Nonces[thr_id]+blockSize, d_partSum1[thr_id], threads);

	// 2
	quark_compactTest_gpu_SCAN<<<thr1,blockSize, 32*sizeof(uint32_t)>>>(d_tempBranch2Nonces[thr_id], 32, d_partSum1[thr_id], h_QuarkFalseFunction[thr_id], threads, startNounce, inpHashes);
	quark_compactTest_gpu_SCAN<<<thr2,blockSize, 32*sizeof(uint32_t)>>>(d_partSum1[thr_id], 32, d_partSum2[thr_id]);
	quark_compactTest_gpu_SCAN<<<1, thr2, 32*sizeof(uint32_t)>>>(d_partSum2[thr_id], (thr2>32) ? 32 : thr2);
	cudaStreamSynchronize(NULL);
	cudaMemcpy(&nrm[1], &(d_partSum2[thr_id])[thr2-1], sizeof(uint32_t), cudaMemcpyDeviceToHost);	
	quark_compactTest_gpu_ADD<<<thr2-1, blockSize>>>(d_partSum1[thr_id]+blockSize, d_partSum2[thr_id], blockSize*thr2);
	quark_compactTest_gpu_ADD<<<thr1-1, blockSize>>>(d_tempBranch2Nonces[thr_id]+blockSize, d_partSum1[thr_id], threads);
	
	// Hier ist noch eine Besonderheit: in d_tempBranch1Nonces sind die element von 1...nrm1 die Interessanten
	// Schritt 3: Scatter
	quark_compactTest_gpu_SCATTER<<<thr1,blockSize,0>>>(d_tempBranch1Nonces[thr_id], d_nonces1, h_QuarkTrueFunction[thr_id], threads, startNounce, inpHashes);
	quark_compactTest_gpu_SCATTER<<<thr1,blockSize,0>>>(d_tempBranch2Nonces[thr_id], d_nonces2, h_QuarkFalseFunction[thr_id], threads, startNounce, inpHashes);
	cudaStreamSynchronize(NULL);
	*/
}

__host__ void quark_compactTest_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable,
											uint32_t *d_nonces1, size_t *nrm1,
											uint32_t *d_nonces2, size_t *nrm2,
											int order)
{
	// Wenn validNonceTable genutzt wird, dann werden auch nur die Nonces betrachtet, die dort enthalten sind
	// "threads" ist in diesem Fall auf die L�nge dieses Array's zu setzen!
	
	quark_compactTest_cpu_dualCompaction(thr_id, threads,
		h_numValid[thr_id], d_nonces1, d_nonces2,
		startNounce, inpHashes, d_validNonceTable);

	hipStreamSynchronize(NULL); // Das original braucht zwar etwas CPU-Last, ist an dieser Stelle aber evtl besser
	*nrm1 = (size_t)h_numValid[thr_id][0];
	*nrm2 = (size_t)h_numValid[thr_id][1];
}

__host__ void quark_compactTest_single_false_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable,
											uint32_t *d_nonces1, size_t *nrm1,
											int order)
{
	// Wenn validNonceTable genutzt wird, dann werden auch nur die Nonces betrachtet, die dort enthalten sind
	// "threads" ist in diesem Fall auf die L�nge dieses Array's zu setzen!

	quark_compactTest_cpu_singleCompaction(thr_id, threads, h_numValid[thr_id], d_nonces1, h_QuarkFalseFunction[thr_id], startNounce, inpHashes, d_validNonceTable);

	hipStreamSynchronize(NULL); // Das original braucht zwar etwas CPU-Last, ist an dieser Stelle aber evtl besser
	*nrm1 = (size_t)h_numValid[thr_id][0];
}
